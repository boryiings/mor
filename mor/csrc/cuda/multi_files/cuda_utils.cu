#include "cuda_utils.cuh"

// Function implementation
int get_num_SMs() {
    int device;
    hipGetDevice(&device);

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device);

    return numSMs;
} 